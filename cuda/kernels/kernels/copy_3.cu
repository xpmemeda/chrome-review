#include "hip/hip_runtime.h"
#include <cstdint>

#include "c10/cuda/CUDAStream.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "torch/all.h"

#include "../module.h"
#include "../support/assert.h"

namespace {

template <int FEATURE_SIZE, int NUM_THREADS>
__global__ void copy_3(  //
    half* dst,           // [n, c, h, w]
    const half* src,     // [n, c, h, w]
    int stride_n,        //
    int stride_c,        //
    int stride_h,        //
    int h,               //
    int w                //
) {
  assert(FEATURE_SIZE == w && "feature_size != w");
  int idx_n = blockIdx.x;
  int idx_c = blockIdx.y;

  constexpr int WARP_SIZE = 32;
  static_assert(NUM_THREADS % WARP_SIZE == 0);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

  for (int idx_h = warp; idx_h < h; idx_h += NUM_WARPS) {
    auto dst_ptr = dst + idx_n * stride_n + idx_c * stride_c + idx_h * stride_h;
    auto src_ptr = src + idx_n * stride_n + idx_c * stride_c + idx_h * stride_h;

    // NOTE: This is a negative example.
    //       Threads within the same warp accessing non-contiguous memory will cause memory coalescing to fail, leading
    //       to performance degradation.
    constexpr int NUM_LOADS = FEATURE_SIZE / WARP_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_LOADS; ++i) {
      const int offset = lane * NUM_LOADS + i;
      *(dst_ptr + offset) = *(src_ptr + offset);
    }
  }
}

void copy_3_launcher(torch::Tensor& dst, const torch::Tensor& src) {
  cr::cr_assert(dst.is_contiguous() && src.is_contiguous(), "not contiguous");
  cr::cr_assert(src.size(2) == 1024, "");
  cr::cr_assert(src.size(3) == 1024, "");

  int n = src.size(0);
  int c = src.size(1);
  int stride_n = src.stride(0);
  int stride_c = src.stride(1);
  int stride_h = src.stride(2);
  dim3 grids(n, c);

  half* dst_ptr = reinterpret_cast<half*>(dst.data_ptr());
  const half* src_ptr = reinterpret_cast<const half*>(src.data_ptr());
  copy_3<1024, 128><<<grids, 128>>>(dst_ptr, src_ptr, stride_n, stride_c, stride_h, 1024, 1024);
}

static cr::Register _([](pybind11::module& m) { m.def("copy_3", &copy_3_launcher); });

}  // namespace
