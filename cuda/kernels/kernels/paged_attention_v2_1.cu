#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <iostream>

#include "c10/cuda/CUDAStream.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "torch/all.h"
#include "torch/csrc/api/include/torch/all.h"

#include "../framework/tensor.h"
#include "../module.h"
#include "../support/assert.h"
#include "../support/exceptions.h"
#include "./attentioncomm/attention_dtypes.h"
#include "./attentioncomm/attention_generic.cuh"
#include "./attentioncomm/attention_utils.cuh"

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace {

class TmpMemoryHolder {
  char* buffer = nullptr;
  size_t allocated_size = 0;

 public:
  template <class T>
  std::tuple<T*, float*, float*> getTmpBuffers(
      int num_seqs, int num_heads, int max_num_partitions, int head_size, hipStream_t stream) {
    size_t nbytes_tmp_out = sizeof(T) * num_seqs * num_heads * max_num_partitions * head_size;
    nbytes_tmp_out = DIVIDE_ROUND_UP(nbytes_tmp_out, 64) * 64;
    size_t nbytes_exp_sums = sizeof(float) * num_seqs * num_heads * max_num_partitions;
    nbytes_exp_sums = DIVIDE_ROUND_UP(nbytes_exp_sums, 64) * 64;
    size_t nbytes_max_logits = sizeof(float) * num_seqs * num_heads * max_num_partitions;
    nbytes_max_logits = DIVIDE_ROUND_UP(nbytes_max_logits, 64) * 64;
    size_t nbytes_required = nbytes_tmp_out + nbytes_exp_sums + nbytes_max_logits;

    if (nbytes_required > allocated_size) {
      cr::check_cuda_err(hipFreeAsync(buffer, stream), "free buffer err");
      cr::check_cuda_err(hipMallocAsync(&buffer, 2 * nbytes_required, stream), "alloc buffer err.");
      allocated_size = 2 * nbytes_required;
    }

    T* tmp_out_ptr = reinterpret_cast<T*>(buffer);
    float* exp_sums_ptr = reinterpret_cast<float*>(buffer + nbytes_tmp_out);
    float* max_logits_ptr = reinterpret_cast<float*>(buffer + nbytes_tmp_out + nbytes_exp_sums);

    return std::make_tuple(tmp_out_ptr, exp_sums_ptr, max_logits_ptr);
  }
};

static TmpMemoryHolder memory_holder;

using namespace vllm;

// Utility function for attention softmax.
template <int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  // Decompose the thread index into warp / lane.
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

  // Compute the sum per warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  // Warp leaders store the data to shared memory.
  if (lane == 0) {
    red_smem[warp] = sum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The warps compute the final sums.
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }

  // Parallel reduction inside the warp.
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  // Broadcast to other threads.
  return __shfl_sync(uint32_t(-1), sum, 0);
}

// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, int HEAD_SIZE, int BLOCK_SIZE, int NUM_THREADS, int PARTITION_SIZE = 0>
__device__ void paged_attention_kernel(        //
    float* __restrict__ exp_sums,              // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,            // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ out,                // [num_seqs, num_heads, max_num_partitions, head_size]
    const scalar_t* __restrict__ q,            // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ k_cache,      // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const scalar_t* __restrict__ v_cache,      // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,                    // [num_heads]
    const float scale,                         //
    const int32_t* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int32_t* __restrict__ context_lens,  // [num_seqs]
    const int max_num_blocks_per_seq,          //
    const float* __restrict__ alibi_slopes,    // [num_heads]
    const int q_stride,                        //
    const int kv_block_stride,                 //
    const int kv_head_stride                   //
) {
  const int seq_idx = blockIdx.y;
  const int partition_idx = blockIdx.z;
  const int max_num_partitions = gridDim.z;
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int context_len = context_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= context_len) {
    // No work to do. Terminate the thread block.
    return;
  }

  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
  const int num_blocks_per_partition = USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_context_blocks;

  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx = USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx = MIN(start_block_idx + num_blocks_per_partition, num_context_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx = MIN(start_token_idx + num_blocks * BLOCK_SIZE, context_len);
  const int num_tokens = end_token_idx - start_token_idx;

  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_THREAD_GROUPS =
      NUM_THREADS / THREAD_GROUP_SIZE;  // Note: This assumes THREAD_GROUP_SIZE divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP = DIVIDE_ROUND_UP(BLOCK_SIZE, WARP_SIZE);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];

  // A vector type to store a part of a key or a query.
  // The vector size is configured in such a way that the threads in a thread group
  // fetch or compute 16 bytes at a time.
  // For example, if the size of a thread group is 4 and the data type is half,
  // then the vector size is 16 / (4 * sizeof(half)) == 2.
  constexpr int VEC_SIZE = MAX(16 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;

  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers.
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in the group
  // has 0, 4, 8, ... th vectors of the query, and the second thread has 1, 5, 9, ...
  // th vectors of the query, and so on.
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
#pragma unroll
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD; i += NUM_THREAD_GROUPS) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    q_vecs[thread_group_offset][i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }
  __syncthreads();  // TODO(naed90): possible speedup if this is replaced with a memory wall right before we use q_vecs

  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 for the softmax logits for better accuracy.
  float* logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  constexpr int x = 16 / sizeof(scalar_t);
  float qk_max = -FLT_MAX;

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  const int32_t* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    // NOTE(woosuk): The block number is stored in int32. However, we cast it to int64
    // because int32 can lead to overflow when this variable is multiplied by large numbers
    // (e.g., kv_block_stride).
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);

    // Load a key to registers.
    // Each thread in a thread group has a different part of the key.
    // For example, if the the thread group size is 4, then the first thread in the group
    // has 0, 4, 8, ... th vectors of the key, and the second thread has 1, 5, 9, ... th
    // vectors of the key, and so on.
    for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      K_vec k_vecs[NUM_VECS_PER_THREAD];

#pragma unroll
      for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        const scalar_t* k_ptr = k_cache + physical_block_number * kv_block_stride + kv_head_idx * kv_head_stride +
                                physical_block_offset * x;
        const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
        const int offset1 = (vec_idx * VEC_SIZE) / x;
        const int offset2 = (vec_idx * VEC_SIZE) % x;
        k_vecs[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
      }

      // Compute dot product.
      // // This includes a reduction across the threads in the same thread group.
      // float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs[thread_group_offset], k_vecs);
      // // Add the ALiBi bias if slopes are given.
      // qk += (alibi_slope != 0) ? alibi_slope * (token_idx - context_len + 1) : 0;

      // 这里计算和原版有一点点不同，wnr的alibi_slope直接加在未scale的S上，参考get_alibi_slope_memref函数
      float qk = Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs[thread_group_offset], k_vecs);
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - context_len + 1) : 0;
      qk *= scale;

      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        const bool mask = token_idx >= context_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        // Update the max value.
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }

  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = qk_max;
  }
  __syncthreads();

  // TODO(woosuk): Refactor this part.
  // Get the max qk value for the sequence.
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  // Broadcast the max qk value to all threads.
  qk_max = VLLM_SHFL_SYNC(qk_max, 0);

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);

  // Compute softmax.
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  // If partitioning is enabled, store the max logit and exp_sum.
  if (USE_PARTITIONING && thread_idx == 0) {
    float* max_logits_ptr =
        max_logits + seq_idx * num_heads * max_num_partitions + head_idx * max_num_partitions + partition_idx;
    *max_logits_ptr = qk_max;
    float* exp_sums_ptr =
        exp_sums + seq_idx * num_heads * max_num_partitions + head_idx * max_num_partitions + partition_idx;
    *exp_sums_ptr = exp_sum;
  }

  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = MIN(16 / sizeof(scalar_t), BLOCK_SIZE);
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using Float_L_vec = typename FloatVec<L_vec>::Type;

  constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
  constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
  constexpr int NUM_ROWS_PER_THREAD = DIVIDE_ROUND_UP(HEAD_SIZE, NUM_ROWS_PER_ITER);

  // NOTE(woosuk): We use FP32 for the accumulator for better accuracy.
  float accs[NUM_ROWS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    accs[i] = 0.f;
  }

  scalar_t zero_value;
  zero(zero_value);
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    // NOTE(woosuk): The block number is stored in int32. However, we cast it to int64
    // because int32 can lead to overflow when this variable is multiplied by large numbers
    // (e.g., kv_block_stride).
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);
    const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
    const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
    L_vec logits_vec;
    from_float(logits_vec, *reinterpret_cast<Float_L_vec*>(logits + token_idx - start_token_idx));

    const scalar_t* v_ptr = v_cache + physical_block_number * kv_block_stride + kv_head_idx * kv_head_stride;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE) {
        const int offset = row_idx * BLOCK_SIZE + physical_block_offset;
        V_vec v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
        if (block_idx == num_context_blocks - 1) {
          // NOTE(woosuk): When v_vec contains the tokens that are out of the context,
          // we should explicitly zero out the values since they may contain NaNs.
          // See https://github.com/vllm-project/vllm/issues/641#issuecomment-1682544472
          scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vec);
#pragma unroll
          for (int j = 0; j < V_VEC_SIZE; j++) {
            v_vec_ptr[j] = token_idx + j < context_len ? v_vec_ptr[j] : zero_value;
          }
        }
        accs[i] += dot(logits_vec, v_vec);
      }
    }
  }

  // Perform reduction within each warp.
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    float acc = accs[i];
#pragma unroll
    for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
      acc += VLLM_SHFL_XOR_SYNC(acc, mask);
    }
    accs[i] = acc;
  }

  // NOTE(woosuk): A barrier is required because the shared memory space for logits
  // is reused for the output.
  __syncthreads();

  // Perform reduction across warps.
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    // Upper warps write to shared memory.
    if (warp_idx >= mid && warp_idx < i) {
      float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[i];
        }
      }
    }
    __syncthreads();

    // Lower warps update the output.
    if (warp_idx < mid) {
      const float* src = &out_smem[warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }

  // Write the final output.
  if (warp_idx == 0) {
    scalar_t* out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
                        head_idx * max_num_partitions * HEAD_SIZE + partition_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        from_float(*(out_ptr + row_idx), accs[i]);
      }
    }
  }
}

// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, int HEAD_SIZE, int BLOCK_SIZE, int NUM_THREADS,
    int PARTITION_SIZE>
__global__ void paged_attention_v2_kernel_1(float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,                                        // [num_seqs, num_heads, max_num_partitions]
    scalar_t* __restrict__ tmp_out,        // [num_seqs, num_heads, max_num_partitions, head_size]
    const scalar_t* __restrict__ q,        // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const scalar_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads, head_size, block_size]
    const int num_kv_heads,                // [num_heads]
    const float scale,
    const int32_t* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int32_t* __restrict__ context_lens,  // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride) {
  paged_attention_kernel<scalar_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS, PARTITION_SIZE>(exp_sums, max_logits, tmp_out, q,
      k_cache, v_cache, num_kv_heads, scale, block_tables, context_lens, max_num_blocks_per_seq, alibi_slopes, q_stride,
      kv_block_stride, kv_head_stride);
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
    int PARTITION_SIZE>
__global__ void paged_attention_v2_reduce_kernel_1(scalar_t* __restrict__ out,  // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,        // [num_seqs, num_heads, max_num_partitions]
    const float* __restrict__ max_logits,      // [num_seqs, num_heads, max_num_partitions]
    const scalar_t* __restrict__ tmp_out,      // [num_seqs, num_heads, max_num_partitions, head_size]
    const int32_t* __restrict__ context_lens,  // [num_seqs]
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int context_len = context_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(context_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    // No need to reduce. Only copy tmp_out to out.
    scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    const scalar_t* tmp_out_ptr =
        tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE + head_idx * max_num_partitions * HEAD_SIZE;
    for (int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
      out_ptr[i] = tmp_out_ptr[i];
    }
    // Terminate the thread block.
    return;
  }

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warp_idx = threadIdx.x / WARP_SIZE;
  const int lane = threadIdx.x % WARP_SIZE;

  // Size: 2 * num_partitions.
  extern __shared__ char shared_mem[];
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // Load max logits to shared memory.
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  const float* max_logits_ptr = max_logits + seq_idx * num_heads * max_num_partitions + head_idx * max_num_partitions;
  float max_logit = -FLT_MAX;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    const float l = max_logits_ptr[i];
    shared_max_logits[i] = l;
    max_logit = fmaxf(max_logit, l);
  }
  __syncthreads();

  // Get the global max logit.
  // Reduce within the warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = max_logit;
  }
  __syncthreads();
  // Reduce across warps.
  max_logit = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  // Broadcast the max value to all threads.
  max_logit = VLLM_SHFL_SYNC(max_logit, 0);

  // Load rescaled exp sums to shared memory.
  float* shared_exp_sums = reinterpret_cast<float*>(shared_mem + sizeof(float) * num_partitions);
  const float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions + head_idx * max_num_partitions;
  float global_exp_sum = 0.0f;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    float l = shared_max_logits[i];
    float rescaled_exp_sum = exp_sums_ptr[i] * expf(l - max_logit);
    global_exp_sum += rescaled_exp_sum;
    shared_exp_sums[i] = rescaled_exp_sum;
  }
  __syncthreads();
  global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.0f, global_exp_sum + 1e-6f);

  // Aggregate tmp_out to out.
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE + head_idx * max_num_partitions * HEAD_SIZE;
  scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
#pragma unroll
  for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
    float acc = 0.0f;
    for (int j = 0; j < num_partitions; ++j) {
      acc += to_float(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] * inv_global_exp_sum;
    }
    from_float(out_ptr[i], acc);
  }
}

#define LAUNCH_PAGED_ATTENTION_V2(HEAD_SIZE)                                                                          \
  paged_attention_v2_kernel_1<T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS, PARTITION_SIZE>                                  \
      <<<grid, block, shared_mem_size, stream>>>(exp_sums_ptr, max_logits_ptr, tmp_out_ptr, query_ptr, key_cache_ptr, \
          value_cache_ptr, num_kv_heads, scale, block_tables_ptr, context_lens_ptr, max_num_blocks_per_seq,           \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride);                                               \
  paged_attention_v2_reduce_kernel_1<T, HEAD_SIZE, NUM_THREADS, PARTITION_SIZE>                                       \
      <<<reduce_grid, block, reduce_shared_mem_size, stream>>>(                                                       \
          out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr, context_lens_ptr, max_num_partitions);

template <typename T, int BLOCK_SIZE, int NUM_THREADS = 128, int PARTITION_SIZE = 512>
void paged_attention_v2_launcher(    //
    cr::Tensor& out,                 // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,         // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,     // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cr::Tensor& value_cache,   // [num_blocks, num_kv_heads, head_size, block_size]
    float scale,                     //
    const cr::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,  // [num_seqs]
    int max_context_len,             //
    const cr::Tensor& alibi_slopes,  // [num_heads]
    hipStream_t stream              //
) {
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;

  int num_seqs = block_tables.size(0);
  int num_heads = query.size(1);
  int num_kv_heads = key_cache.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  int max_num_partitions = DIVIDE_ROUND_UP(max_context_len, PARTITION_SIZE);
  int logits_size = PARTITION_SIZE * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);

  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  cr::cr_assert(head_size % thread_group_size == 0, "pagedattention internal error");

  auto alibi_slopes_ptr = alibi_slopes.data<float>();
  auto out_ptr = out.data<T>();
  auto [tmp_out_ptr, exp_sums_ptr, max_logits_ptr] =
      memory_holder.getTmpBuffers<T>(num_seqs, num_heads, max_num_partitions, head_size, stream);
  auto query_ptr = query.data<T>();
  auto key_cache_ptr = key_cache.data<T>();
  auto value_cache_ptr = value_cache.data<T>();
  auto block_tables_ptr = block_tables.data<int32_t>();
  auto context_lens_ptr = context_lens.data<int32_t>();

  dim3 grid(num_heads, num_seqs, max_num_partitions);
  int shared_mem_size = std::max(logits_size, outputs_size);
  dim3 reduce_grid(num_heads, num_seqs);
  int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);
  dim3 block(NUM_THREADS);
  switch (head_size) {
    case 64:
      LAUNCH_PAGED_ATTENTION_V2(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V2(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V2(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V2(112);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V2(128);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V2(256);
      break;
    default:
      cr::cr_assert(false, "Unsupported head size: " + std::to_string(head_size));
      break;
  }
}

#define CALL_V2_LAUNCHER(T, BLOCK_SIZE)       \
  paged_attention_v2_launcher<T, BLOCK_SIZE>( \
      out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);

#define CALL_V2_LAUNCHER_BLOCK_SIZE(T)                                               \
  switch (block_size) {                                                              \
    case 8:                                                                          \
      CALL_V2_LAUNCHER(T, 8);                                                        \
      break;                                                                         \
    case 16:                                                                         \
      CALL_V2_LAUNCHER(T, 16);                                                       \
      break;                                                                         \
    case 32:                                                                         \
      CALL_V2_LAUNCHER(T, 32);                                                       \
      break;                                                                         \
    default:                                                                         \
      cr::cr_assert(false, "Unsupported block size: " + std::to_string(block_size)); \
      break;                                                                         \
  }

void paged_attention_v2_1_internal(  //
    cr::Tensor& out,                 // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,         // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,     // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const cr::Tensor& value_cache,   // [num_blocks, num_kv_heads, head_size, block_size]
    float scale,                     //
    const cr::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,  // [num_seqs]
    int max_context_len,             //
    const cr::Tensor& alibi_slopes,  // [num_heads]
    hipStream_t stream              //
) {
  int block_size = key_cache.size(3);
  CALL_V2_LAUNCHER_BLOCK_SIZE(uint16_t);
}

void paged_attention_v2_1(                            //
    torch::Tensor& out,                               // [num_seqs, num_heads, head_size]
    const torch::Tensor& query,                       // [num_seqs, num_heads, head_size]
    const torch::Tensor& key_cache,                   // [num_blocks, num_kv_heads, head_size/x, block_size, x]
    const torch::Tensor& value_cache,                 // [num_blocks, num_kv_heads, head_size, block_size]
    float scale,                                      //
    const torch::Tensor& block_tables,                // [num_seqs, max_num_blocks_per_seq]
    const torch::Tensor& context_lens,                // [num_seqs]
    int32_t max_context_len,                          //
    const c10::optional<torch::Tensor>& alibi_slopes  // [num_heads]
) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto cr_out = cr::Tensor::referenceFromTorchTensor(out);
  auto cr_query = cr::Tensor::referenceFromTorchTensor(query);
  auto cr_key_cache = cr::Tensor::referenceFromTorchTensor(key_cache);
  auto cr_value_cache = cr::Tensor::referenceFromTorchTensor(value_cache);
  auto cr_block_tables = cr::Tensor::referenceFromTorchTensor(block_tables);
  auto cr_context_lens = cr::Tensor::referenceFromTorchTensor(context_lens);
  auto cr_alibi_slopes = alibi_slopes ? cr::Tensor::referenceFromTorchTensor(*alibi_slopes) : cr::Tensor();
  paged_attention_v2_1_internal(cr_out, cr_query, cr_key_cache, cr_value_cache, scale, cr_block_tables, cr_context_lens,
      max_context_len, cr_alibi_slopes, stream);
}

static cr::Register _(
    [](pybind11::module& m) { m.def("paged_attention_v2_1", &paged_attention_v2_1, "Layout_K.VLLM; Layout_V.VLLM"); });

}  // namespace
