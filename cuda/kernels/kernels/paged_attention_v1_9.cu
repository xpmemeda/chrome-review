#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdint>
#include <iostream>

#include "c10/cuda/CUDAStream.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "pybind11/pybind11.h"
#include "torch/all.h"
#include "torch/csrc/api/include/torch/all.h"

#include "../framework/tensor.h"
#include "../module.h"
#include "../support/assert.h"
#include "../support/exceptions.h"
#include "./attentioncomm/attention_dtypes.h"
#include "./attentioncomm/attention_generic.cuh"
#include "./attentioncomm/attention_utils.cuh"
#include "./attentioncomm/crutils.cuh"

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

namespace {

using namespace vllm;
using namespace cr;

// Grid: (num_heads, num_seqs). One "BLOCK" works on one "SEQ-HEAD".
template <typename scalar_t, int HEAD_SIZE, int BLOCK_SIZE,
    int NUM_THREADS>
__global__ void single_query_cached_kv_attention_kernel_9(  //
    scalar_t* __restrict__ out,                             // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ q,                         // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ k_cache,                   // [num_blocks, num_kv_heads, block_size, head_size]
    const scalar_t* __restrict__ v_cache,                   // [num_blocks, num_kv_heads, block_size, head_size]
    const int num_heads_kv,                                 //
    const float scale,                                      //
    const int32_t* __restrict__ block_tables,               // [num_seqs, max_num_blocks_per_seq]
    const int32_t* __restrict__ context_lens,               // [num_seqs]
    const int max_num_blocks_per_seq,                       //
    const float* __restrict__ alibi_slopes,                 // [num_heads]
    const int q_stride,                                     //
    const int kv_block_stride,                              //
    const int kv_head_stride                                //
) {
  static_assert(WARP_SIZE >= BLOCK_SIZE && WARP_SIZE % BLOCK_SIZE == 0);
  static_assert(NUM_THREADS % WARP_SIZE == 0);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;

  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int kv_head_idx = head_idx / (num_heads / num_heads_kv);
  const int seq_idx = blockIdx.y;
  const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];
  const int32_t* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  const int context_len = context_lens[seq_idx];
  const int num_blocks = (context_len + BLOCK_SIZE - 1) / BLOCK_SIZE;

  extern __shared__ char shared_mem[];
  float* logits = reinterpret_cast<float*>(shared_mem);
  __shared__ float red_smem[2 * NUM_WARPS];

  using LoadQAndGemmQk = LoadQAndGemmQk_CR4<scalar_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>;
  using Softmax = Softmax<HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>;
  using GemmPvAndStoreO = GemmPvAndStoreO_CR1<scalar_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>;

  constexpr int N = HEAD_SIZE / WARP_SIZE;
  using vec_t = typename Vec<scalar_t, N>::Type;
  __shared__ vec_t smem_k_vecs[LoadQAndGemmQk::NUM_STAGES * NUM_WARPS][WARP_SIZE];
  auto load_q_and_gemm_qk = LoadQAndGemmQk(q, &smem_k_vecs[0][0], k_cache, block_table, logits, red_smem, seq_idx,
      head_idx, kv_head_idx, q_stride, kv_block_stride, kv_head_stride, context_len, num_blocks, scale, alibi_slope);
  load_q_and_gemm_qk.loadQ();
  float qk_max = load_q_and_gemm_qk.gemmQk();

  Softmax::doSoftmax(logits, red_smem, qk_max, context_len);

  scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
  auto gemm_pv_and_store_o = GemmPvAndStoreO(out_ptr, v_cache, block_table, logits, seq_idx, head_idx, kv_head_idx,
      num_heads, num_blocks, kv_block_stride, kv_head_stride, context_len);
  gemm_pv_and_store_o.gemmPv();
  gemm_pv_and_store_o.storeO();
}

template <typename... Args>
void single_query_cached_kv_attention_launcher(  //
    cr::Tensor& out,                             // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,                     // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    const cr::Tensor& value_cache,               // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                 //
    const cr::Tensor& block_tables,              // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,              // [num_seqs]
    int max_context_len,                         //
    const cr::Tensor& alibi_slopes,              // [num_heads]
    hipStream_t stream                          //
);

template <int BLOCK_SIZE, int HEAD_SIZE, int NUM_THREADS>
void single_query_cached_kv_attention_launcher(  //
    cr::Tensor& out,                             // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,                     // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    const cr::Tensor& value_cache,               // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                 //
    const cr::Tensor& block_tables,              // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,              // [num_seqs]
    int max_context_len,                         //
    const cr::Tensor& alibi_slopes,              // [num_heads]
    hipStream_t stream                          //
) {
  using T = uint16_t;

  int num_seqs = block_tables.size(0);
  int num_heads = query.size(1);
  int num_heads_kv = key_cache.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  auto alibi_slopes_ptr = alibi_slopes.data<float>();
  auto out_ptr = out.data<T>();
  auto query_ptr = query.data<T>();
  auto key_cache_ptr = key_cache.data<T>();
  auto value_cache_ptr = value_cache.data<T>();
  auto block_tables_ptr = block_tables.data<int32_t>();
  auto context_lens_ptr = context_lens.data<int32_t>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_context_len = ((max_context_len + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_context_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);
  int shared_mem_size = std::max(logits_size, outputs_size);

  auto kernel = &single_query_cached_kv_attention_kernel_9<T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>;

  dim3 grid(num_heads, num_seqs);
  dim3 block(NUM_THREADS);
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
  kernel<<<grid, block, shared_mem_size, stream>>>(out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_heads_kv,
      scale, block_tables_ptr, context_lens_ptr, max_num_blocks_per_seq, alibi_slopes_ptr, q_stride, kv_block_stride,
      kv_head_stride);
}

template <int BLOCK_SIZE, int HEAD_SIZE>
void single_query_cached_kv_attention_launcher(  //
    cr::Tensor& out,                             // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,                     // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    const cr::Tensor& value_cache,               // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                 //
    const cr::Tensor& block_tables,              // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,              // [num_seqs]
    int max_context_len,                         //
    const cr::Tensor& alibi_slopes,              // [num_heads]
    hipStream_t stream                          //
) {
  constexpr int NUM_THREADS = 128;
  single_query_cached_kv_attention_launcher<BLOCK_SIZE, HEAD_SIZE, NUM_THREADS>(
      out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
}

template <int BLOCK_SIZE>
void single_query_cached_kv_attention_launcher(  //
    cr::Tensor& out,                             // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,                     // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    const cr::Tensor& value_cache,               // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                 //
    const cr::Tensor& block_tables,              // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,              // [num_seqs]
    int max_context_len,                         //
    const cr::Tensor& alibi_slopes,              // [num_heads]
    hipStream_t stream                          //
) {
  int head_size = query.size(2);

  switch (head_size) {
    case 64: {
      single_query_cached_kv_attention_launcher<BLOCK_SIZE, 64>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    case 128: {
      single_query_cached_kv_attention_launcher<BLOCK_SIZE, 128>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    case 256: {
      single_query_cached_kv_attention_launcher<BLOCK_SIZE, 256>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    default:
      throw std::runtime_error("unsupported head size: " + std::to_string(head_size));
      break;
  }
}

template <>
void single_query_cached_kv_attention_launcher(  //
    cr::Tensor& out,                             // [num_seqs, num_heads, head_size]
    const cr::Tensor& query,                     // [num_seqs, num_heads, head_size]
    const cr::Tensor& key_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    const cr::Tensor& value_cache,               // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                 //
    const cr::Tensor& block_tables,              // [num_seqs, max_num_blocks_per_seq]
    const cr::Tensor& context_lens,              // [num_seqs]
    int max_context_len,                         //
    const cr::Tensor& alibi_slopes,              // [num_heads]
    hipStream_t stream                          //
) {
  int block_size = key_cache.size(2);

  switch (block_size) {
    case 8: {
      single_query_cached_kv_attention_launcher<8>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    case 16: {
      single_query_cached_kv_attention_launcher<16>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    case 32: {
      single_query_cached_kv_attention_launcher<32>(
          out, query, key_cache, value_cache, scale, block_tables, context_lens, max_context_len, alibi_slopes, stream);
      break;
    }
    default:
      throw std::runtime_error("unsupported block size: " + std::to_string(block_size));
      break;
  }
}

void decode_attn_9(                                   //
    torch::Tensor& out,                               // [num_seqs, num_heads, head_size]
    const torch::Tensor& query,                       // [num_seqs, num_heads, head_size]
    const torch::Tensor& key_cache,                   // [num_blocks, num_heads_kv, block_size, head_size]
    const torch::Tensor& value_cache,                 // [num_blocks, num_heads_kv, block_size, head_size]
    float scale,                                      //
    const torch::Tensor& block_tables,                // [num_seqs, max_num_blocks_per_seq]
    const torch::Tensor& context_lens,                // [num_seqs]
    int max_context_len,                              //
    const c10::optional<torch::Tensor>& alibi_slopes  // [num_heads]
) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  do {
    auto cr_out = cr::Tensor::referenceFromTorchTensor(out);
    auto cr_query = cr::Tensor::referenceFromTorchTensor(query);
    auto cr_key_cache = cr::Tensor::referenceFromTorchTensor(key_cache);
    auto cr_value_cache = cr::Tensor::referenceFromTorchTensor(value_cache);
    auto cr_block_tables = cr::Tensor::referenceFromTorchTensor(block_tables);
    auto cr_context_lens = cr::Tensor::referenceFromTorchTensor(context_lens);
    auto cr_alibi_slopes = alibi_slopes ? cr::Tensor::referenceFromTorchTensor(*alibi_slopes) : cr::Tensor();
    single_query_cached_kv_attention_launcher(cr_out, cr_query, cr_key_cache, cr_value_cache, scale, cr_block_tables,
        cr_context_lens, max_context_len, cr_alibi_slopes, stream);
  } while (0);
  return;
}

static cr::Register _(
    [](pybind11::module& m) { m.def("paged_attention_v1_9", &decode_attn_9, "Layout_K.CR; Layout_V.CR"); });

}  // namespace
