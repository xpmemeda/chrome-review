#include "conv.h"

#include <cassert>
#include <iostream>

#include "c10/cuda/CUDAStream.h"
#include "hip/hip_runtime.h"
#include "hipDNN.h"

#include "../support/assert.h"
#include "../support/cudnntraits.h"
#include "../support/exceptions.h"
#include "../support/utils.h"

namespace cr {

namespace {

template <typename T>
void conv3d_v1_kernel(const T* input, const T* filter, T* output,  //
    const int* input_dims, const int* input_strides,               // input
    const int* filter_dims, const int* filter_strides,             // filter
    const int* output_dims, const int* output_strides,             // output
    hipdnnHandle_t cudnn_handle,                                    //
    const int* conv_paddings, const int* conv_strides, const int* conv_dilates, int group) {
  hipdnnTensorDescriptor_t input_desc;
  check_cuda_err(hipdnnCreateTensorDescriptor(&input_desc));
  check_cuda_err(
      hipdnnSetTensorNdDescriptor(input_desc, CudnnDataTypeTrait<T>::data_type, 5, input_dims, input_strides));

  hipdnnFilterDescriptor_t filter_desc;
  check_cuda_err(hipdnnCreateFilterDescriptor(&filter_desc));
  check_cuda_err(
      hipdnnSetFilterNdDescriptor(filter_desc, CudnnDataTypeTrait<T>::data_type, HIPDNN_TENSOR_NCHW, 5, filter_dims));

  hipdnnTensorDescriptor_t output_desc;
  check_cuda_err(hipdnnCreateTensorDescriptor(&output_desc));
  check_cuda_err(
      hipdnnSetTensorNdDescriptor(output_desc, CudnnDataTypeTrait<T>::data_type, 5, output_dims, output_strides));

  hipdnnConvolutionDescriptor_t conv_op_desc;
  check_cuda_err(hipdnnCreateConvolutionDescriptor(&conv_op_desc));
  check_cuda_err(hipdnnSetConvolutionNdDescriptor(
      conv_op_desc, 3, conv_paddings, conv_strides, conv_dilates, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  check_cuda_err(hipdnnSetConvolutionGroupCount(conv_op_desc, group));

  const float alpha = 1.0f, beta = 0.0f;
  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  do {
    hipdnnConvolutionFwdAlgoPerf_t perf;
    check_cuda_err(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn_handle, input_desc, filter_desc, conv_op_desc, output_desc, 1, nullptr, &perf));
    algo = perf.algo;
  } while (0);

  size_t workspace_size = 0;
  void* workspace = nullptr;
  check_cuda_err(hipdnnGetConvolutionForwardWorkspaceSize(
      cudnn_handle, input_desc, filter_desc, conv_op_desc, output_desc, algo, &workspace_size));
  if (workspace_size > 0) {
    check_cuda_err(hipMalloc(&workspace, workspace_size));
  }

  check_cuda_err(hipdnnConvolutionForward(cudnn_handle, &alpha, input_desc, input, filter_desc, filter, conv_op_desc,
      algo, workspace, workspace_size, &beta, output_desc, output));

  if (workspace_size > 0) {
    check_cuda_err(hipFree(workspace));
  }

  check_cuda_err(hipdnnDestroyTensorDescriptor(input_desc));
  check_cuda_err(hipdnnDestroyFilterDescriptor(filter_desc));
  check_cuda_err(hipdnnDestroyTensorDescriptor(output_desc));
  check_cuda_err(hipdnnDestroyConvolutionDescriptor(conv_op_desc));
}

}  // namespace

void conv_v1(torch::Tensor& input, torch::Tensor& filter, torch::Tensor& output, int group,
    const std::vector<int>& paddings, const std::vector<int>& strides, const std::vector<int>& dilates) {
  std::vector<int> input_dims;
  std::vector<int> input_strides;
  std::vector<int> filter_dims;
  std::vector<int> filter_strides;
  std::vector<int> output_dims;
  std::vector<int> output_strides;
  int rank = input.ndimension();
  for (int i = 0; i < rank; ++i) {
    input_dims.push_back(input.size(i));
    input_strides.push_back(input.stride(i));
    filter_dims.push_back(filter.size(i));
    filter_strides.push_back(filter.stride(i));
    output_dims.push_back(output.size(i));
    output_strides.push_back(output.stride(i));
  }
  int empty_ndim = 5 - rank;
  input_dims.insert(input_dims.begin() + 2, empty_ndim, 1);
  input_strides.insert(input_strides.begin() + 2, empty_ndim, 0);
  filter_dims.insert(filter_dims.begin() + 2, empty_ndim, 1);
  filter_strides.insert(filter_strides.begin() + 2, empty_ndim, 0);
  output_dims.insert(output_dims.begin() + 2, empty_ndim, 1);
  output_strides.insert(output_strides.begin() + 2, empty_ndim, 0);

  std::vector<int> conv_paddings = paddings;
  std::vector<int> conv_strides = strides;
  std::vector<int> conv_dilates = dilates;
  conv_paddings.insert(conv_paddings.begin(), empty_ndim, 0);
  conv_strides.insert(conv_strides.begin(), empty_ndim, 1);
  conv_dilates.insert(conv_dilates.begin(), empty_ndim, 1);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  hipdnnHandle_t cudnn_handle;
  check_cuda_err(hipdnnCreate(&cudnn_handle));
  check_cuda_err(hipdnnSetStream(cudnn_handle, stream));

  print_vec("input_dims: ", input_dims);
  print_vec("input_strides: ", input_strides);
  print_vec("filter_dims: ", filter_dims);
  print_vec("filter_strides: ", filter_strides);
  print_vec("output_dims: ", output_dims);
  print_vec("output_strides: ", output_strides);
  print_vec("conv_paddings: ", conv_paddings);
  print_vec("conv_strides: ", conv_strides);
  print_vec("conv_dilates: ", conv_dilates);
  printf("group: %d\n", group);

  if (input.dtype() == torch::ScalarType::Half) {
    using T = half;
    auto input_ptr = reinterpret_cast<T*>(input.data_ptr());
    auto kernel_ptr = reinterpret_cast<T*>(filter.data_ptr());
    auto output_ptr = reinterpret_cast<T*>(output.data_ptr());
    conv3d_v1_kernel<T>(input_ptr, kernel_ptr, output_ptr, input_dims.data(), input_strides.data(), filter_dims.data(),
        filter_strides.data(), output_dims.data(), output_strides.data(), cudnn_handle, conv_paddings.data(),
        conv_strides.data(), conv_dilates.data(), group);
  } else if (input.dtype() == torch::ScalarType::Float) {
    using T = float;
    auto input_ptr = reinterpret_cast<T*>(input.data_ptr());
    auto kernel_ptr = reinterpret_cast<T*>(filter.data_ptr());
    auto output_ptr = reinterpret_cast<T*>(output.data_ptr());
    conv3d_v1_kernel<T>(input_ptr, kernel_ptr, output_ptr, input_dims.data(), input_strides.data(), filter_dims.data(),
        filter_strides.data(), output_dims.data(), output_strides.data(), cudnn_handle, conv_paddings.data(),
        conv_strides.data(), conv_dilates.data(), group);
  } else {
    cr::cr_assert(false, "invalid dtype");
  }

  check_cuda_err(hipdnnDestroy(cudnn_handle));
}

}  // namespace cr
